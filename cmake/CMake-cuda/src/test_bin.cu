#include "hip/hip_runtime.h"

#include <external_dependency.h>

__global__ void kernel(unsigned int *in, unsigned int *out) {

  // Note this kernel doesn't do anything. 
  // It's just filler to demonstrate how to setup cmake.

  for (unsigned int i=0;i<blockDim.x;++i) {
    // /*const*/ unsigned int thread = threadIdx.x;
    out[threadIdx.x] = in[threadIdx.x] * in[threadIdx.x];
  }
};






