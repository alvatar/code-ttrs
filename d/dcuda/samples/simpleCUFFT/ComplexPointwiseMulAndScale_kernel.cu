
#include <hip/hip_runtime.h>

// float2 addition
__device__ inline float2 ComplexAdd(float2 a, float2 b) {
	float2 c;
	c.x = a.x + b.x;
	c.y = a.y + b.y;
	return c;
}

// float2 scale
__device__ inline float2 ComplexScale(float2 a, float s) {
	float2 c;
	c.x = s * a.x;
	c.y = s * a.y;
	return c;
}

// float2 multiplication
__device__ inline float2 ComplexMul(float2 a, float2 b) {
	float2 c;
	c.x = a.x * b.x - a.y * b.y;
	c.y = a.x * b.y + a.y * b.x;
	return c;
}

// float2 pointwise multiplication
extern "C" __global__ void ComplexPointwiseMulAndScale(float2* a, const float2* b, int size, float scale) {
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = threadID; i < size; i += numThreads)
		a[i] = ComplexScale(ComplexMul(a[i], b[i]), scale);     
} 

